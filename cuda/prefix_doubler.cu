#include "hip/hip_runtime.h"
#include <string>
#include <iostream>
#include "cub-1.8.0/hipcub/hipcub.hpp"


#define NUM_BLOCKS 2
#define NUM_THREADS_PER_BLOCK 4


struct Max_without_branching
{
    template <typename T>
    HIPCUB_RUNTIME_FUNCTION __forceinline__ __device__
    T operator()(const T &x, const T &y) const {
        return (x ^ ((x ^ y) & -(x < y)));
    }
};

struct Compare_four_chars
{
public:  
    Compare_four_chars(int* _text) : text(_text) {};
    int* text;
    template <typename index>
    HIPCUB_RUNTIME_FUNCTION __forceinline__ __device__
    bool operator()(const index &x, const index &y) const {
        return text[x]<text[y];
    }
};

//Quick and dirty version, which packs four chars in one int
void word_packing(const char* chars, int* result, int n) {

    typedef unsigned char u8; 
    for(int i = 0; i<n-3 ;++i) {
        result[i] = ((u8)chars[i] << 24) | ((u8)chars[i+1] << 16) | ((u8)chars[i+2] << 8) | (u8)chars[i+3];
    }
    result[n-3] = ((u8)chars[n-3] << 24) | ((u8)chars[n-2] << 16) | ((u8)chars[n-1] << 8);
    result[n-2] = ((u8)chars[n-2] << 24) | ((u8)chars[n-1] << 16);
    result[n-1] = ((u8)chars[n-1] << 24);

}

/*
    Init SA on GPU. Every GPU thread writes his index into SA,
    then jumps stride size until end is reached
*/
__global__
static void initialize_sa_gpu(int n, int*  sa) {

    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < n; i+=stride) {
        sa[i] = i;
    }

}

/*
    Copies one array into another by using GPU threads
    Maybe use memcpy?
*/
__global__
static void copy_to_array(int* in, int* out, int n) {

    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < n; i+=stride) {
        in[i] = out[i];
    }

}

/*
    Sorts SA according to text using the CUB Radixsort
*/
static void inital_sorting(int* text, int* sa, int* aux, int n) {

     //Actual values
    int  *keys_out;     // e.g., [        ...        ]


    // Allocate Unified Memory – accessible from CPU or GPU
    hipMallocManaged(&keys_out, n*sizeof(int));


    // Determine temporary device storage requirements
    void     *d_temp_storage = NULL;
    size_t   temp_storage_bytes = 0;

    hipcub::DeviceRadixSort::SortPairs(d_temp_storage, temp_storage_bytes,
        text, keys_out, sa, aux, n);
    // Allocate temporary storage
    hipMalloc(&d_temp_storage, temp_storage_bytes);


    // Run sorting operation
    hipcub::DeviceRadixSort::SortPairs(d_temp_storage, temp_storage_bytes,
        text, keys_out, sa, aux, n);


    hipDeviceSynchronize();

    copy_to_array<<<NUM_BLOCKS,NUM_THREADS_PER_BLOCK>>>(sa,aux,n);
}


/*
    Calculates inclusive prefix sum on GPU using the provided CUB Method
*/
template <typename OP>
void prefix_sum_cub_inclusive(int* array, OP op, int n)
{
    //Indices
    int  *values_out;   // e.g., [        ...        ]

    // Allocate Unified Memory – accessible from CPU or GPU
    hipMallocManaged(&values_out, n*sizeof(int));

    // Determine temporary device storage requirements
    void     *d_temp_storage = NULL;
    size_t   temp_storage_bytes = 0;
    
    hipcub::DeviceScan::InclusiveScan(d_temp_storage, temp_storage_bytes, array, values_out,op, n);
    // Allocate temporary storage
    hipMalloc(&d_temp_storage, temp_storage_bytes);
    // Run exclusive prefix sum
    hipcub::DeviceScan::InclusiveScan(d_temp_storage, temp_storage_bytes, array, values_out,op, n);

    hipDeviceSynchronize();

    copy_to_array<<<NUM_BLOCKS,NUM_THREADS_PER_BLOCK>>>(array,values_out,n);


}
/*
    Auxiliary function for initializing ISA
    Computes inital aux array, with index if own value other to predecessor, else 0
*/
template <typename Comp>
__global__
void fill_aux_for_isa(int* sa, int* aux, int n, Comp comp) {

    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    if(index == 0) {
        aux[0]=0;
    }

    for (int i = index+1; i < n; i+=stride) {
        aux[i] = i * (comp(sa[i - 1], sa[i]) != 0);
    }
}
/*
    Auxiliary function for initializing ISA
    writes aux in ISA
*/
__global__
void scatter_to_isa(int* isa, int* aux,int* sa, int n) {

    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    //Maybe TODO: Avoid Bank Conflicts
    for (int i = index; i < n; i+=stride) {
        isa[sa[i]]=aux[i];
    }
}

/*
    Init ISA with prefix sum method
*/
template <typename Comp>
void initialize_isa(int* isa, int* sa, int* aux, int n, Comp comp) {

    fill_aux_for_isa<<<NUM_BLOCKS,NUM_THREADS_PER_BLOCK>>>(sa,aux,n, comp);

    hipDeviceSynchronize();

    Max_without_branching max;

    prefix_sum_cub_inclusive(aux,max, n);

    hipDeviceSynchronize();

    scatter_to_isa<<<NUM_BLOCKS,NUM_THREADS_PER_BLOCK>>>(isa,aux,sa,n);

}


static void prefix_doubling_gpu(int* gpu_text, int* out_sa, int n) {
    
    //additional arrays
    int* sa;
    int* isa_container;
    int* aux_container;

    //allocate additional arrays directly on GPU
    hipMallocManaged(&sa, n*sizeof(int));
    hipMallocManaged(&isa_container, n*sizeof(int));
    hipMallocManaged(&aux_container, n*sizeof(int));
    hipDeviceSynchronize();

    //Fill SA 
    initialize_sa_gpu<<<NUM_BLOCKS,NUM_THREADS_PER_BLOCK>>>(n, sa);
    hipDeviceSynchronize();

    //Sort by four characters
    inital_sorting(gpu_text, sa, aux_container, n);
    hipDeviceSynchronize();

    std::cout<<"SA: ";
    for(int i = 0; i<n; ++i) {
        std::cout<<sa[i]<<", ";
    }
    std::cout<<std::endl;

    //Init ISA with group numbers according to initial sorting
    Compare_four_chars comp(gpu_text);
    initialize_isa(out_sa, sa, aux_container, n, comp);
    hipDeviceSynchronize();

    std::cout<<std::endl;
    std::cout<<"ISA: ";
    for(int i = 0 ; i< n ; ++i) {
        std::cout<<out_sa[sa[i]]<<", ";
    }
    std::cout<<std::endl;
    
    int h = 4;

/*
    phase.split("Mark singletons");
    mark_singletons(sa, isa);
    phase.split("Loop Initialization");

    // std::cout << "isa: " << isa << std::endl;
    size_t size = sa.size();
    size_t s = 0;

    auto tuple_container =
        util::make_container<std::tuple<sa_index, sa_index, sa_index>>(
            size);
    util::span<std::tuple<sa_index, sa_index, sa_index>> tuples;
    compare_tuples<sa_index> cmp;
    while (size > 0) {
        phase.split("Iteration");
        aux = util::span<sa_index>(aux_container).slice(0, size);
        tuples = tuple_container.slice(0, size);

        //s = create_tuples<sa_index>(tuples.slice(0, size), size, h, sa, isa);
        s = create_tuples_parallel<sa_index>(tuples.slice(0, size),
                size, h, sa, isa, aux);
        //std::cout << "Elements left: " << size << std::endl;

        // std::cout << "Next size: " << s << std::endl;
        // Skip all operations till size gets its new size, if this
        // iteration contains no tuples
        if (s > 0) {
            tuples = tuples.slice(0, s);
            aux = util::span<sa_index>(aux).slice(0, s);
            // std::cout << "Sorting tuples." << std::endl;
            cmp = compare_tuples<sa_index>(tuples);
            util::sort::std_par_stable_sort(tuples, cmp);
            sa = sa.slice(0, s);
            update_ranks_prefixsum(tuples, aux);
            // std::cout << "Writing new order to sa." << std::endl;
            for (size_t i = 0; i < s; ++i) {
                sa[i] = std::get<0>(tuples[i]);
            }

            for (size_t i = 0; i < s; ++i) {

                isa[std::get<0>(tuples[i])] =
                    aux[i]; // std::get<1>(tuples[i]);
            }
            mark_singletons(sa, isa);
        }
        size = s;
        h = 2 * h;
    }
    phase.split("Write out SA");
    for (size_t i = 0; i < out_sa.size(); ++i) {
        out_sa[isa[i] ^ utils<sa_index>::NEGATIVE_MASK] = i;
    }
    */
}
int main()
{
    std::string text_str = "caabaccaabacaa";
    const char* text = text_str.c_str();
    int n = text_str.size()+1;
    std::cout<<"n: "<<n<<std::endl;


    int* packed_text;
    packed_text = (int *) malloc(n*sizeof(int));
    //Pack text, so you can compare four chars at once
    word_packing(text, packed_text, n);

    //GPU arrays
    int* gpu_text;
    int* out_sa;
    hipMallocManaged(&gpu_text, n*sizeof(int));
    //Copy text to GPU
    memset(gpu_text, 0, n*sizeof(int));
    hipMemcpy(gpu_text, packed_text, n*sizeof(int), hipMemcpyHostToDevice);  
    hipMallocManaged(&out_sa, n*sizeof(int));
    hipDeviceSynchronize();


    prefix_doubling_gpu(gpu_text, out_sa, n);

    return 0;
}
