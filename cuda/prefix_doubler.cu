#include "hip/hip_runtime.h"
#include <string>
#include <iostream>
#include "cub-1.8.0/hipcub/hipcub.hpp"


#define NUM_BLOCKS 2
#define NUM_THREADS_PER_BLOCK 4


struct Max_without_branching
{
    template <typename T>
    HIPCUB_RUNTIME_FUNCTION __forceinline__ __device__
    T operator()(const T &x, const T &y) const {
        return (x ^ ((x ^ y) & -(x < y)));
    }
};

struct Compare_first_char
{
public:  
    Compare_first_char(const char* _text) : text(_text) {};
    const char* text;
    template <typename index>
    HIPCUB_RUNTIME_FUNCTION __forceinline__ __device__
    bool operator()(const index &x, const index &y) const {
        return text[x]<text[y];
    }
};

    __global__
    static void initialize_sa_gpu(int n, int*  sa) {

        int index = blockIdx.x * blockDim.x + threadIdx.x;
        int stride = blockDim.x * gridDim.x;
        for (int i = index; i < n; i+=stride) {
            sa[i] = i;
        }

    }

    __global__
    static void copy_to_array(int* in, int* out, int n) {

        int index = blockIdx.x * blockDim.x + threadIdx.x;
        int stride = blockDim.x * gridDim.x;
        for (int i = index; i < n; i+=stride) {
            in[i] = out[i];
        }

    }


    static void inital_sorting(char* text, int* sa, int* aux, int n) {


     //Actual values
    char  *keys_out;     // e.g., [        ...        ]


    // Allocate Unified Memory – accessible from CPU or GPU
    hipMallocManaged(&keys_out, n*sizeof(char));


    // Determine temporary device storage requirements
    void     *d_temp_storage = NULL;
    size_t   temp_storage_bytes = 0;

    hipcub::DeviceRadixSort::SortPairs(d_temp_storage, temp_storage_bytes,
        text, keys_out, sa, aux, n);
    // Allocate temporary storage
    hipMalloc(&d_temp_storage, temp_storage_bytes);


    // Run sorting operation
    hipcub::DeviceRadixSort::SortPairs(d_temp_storage, temp_storage_bytes,
        text, keys_out, sa, aux, n);


    hipDeviceSynchronize();

    


    }

    template <typename OP>
void prefix_sum_cub_inclusive(int* array, OP op, int n)
{
        //Indices
        int  *values_out;   // e.g., [        ...        ]

    
    
        // Allocate Unified Memory – accessible from CPU or GPU
        hipMallocManaged(&values_out, n*sizeof(int));
    
        // Determine temporary device storage requirements
        void     *d_temp_storage = NULL;
        size_t   temp_storage_bytes = 0;
        
        hipcub::DeviceScan::InclusiveScan(d_temp_storage, temp_storage_bytes, array, values_out,op, n);
        // Allocate temporary storage
        hipMalloc(&d_temp_storage, temp_storage_bytes);
        // Run exclusive prefix sum
        hipcub::DeviceScan::InclusiveScan(d_temp_storage, temp_storage_bytes, array, values_out,op, n);

        hipDeviceSynchronize();

        copy_to_array<<<NUM_BLOCKS,NUM_THREADS_PER_BLOCK>>>(array,values_out,n);


}
    template <typename Comp>
    __global__
    void fill_aux_for_isa(int* sa, int* aux, int n, Comp comp) {

        int index = blockIdx.x * blockDim.x + threadIdx.x;
        int stride = blockDim.x * gridDim.x;

        if(index == 0) {
            aux[0]=0;
        }

        for (int i = index+1; i < n; i+=stride) {
            aux[i] = i * (comp(sa[i - 1], sa[i]) != 0);
        }
    }
    __global__
    void scatter_to_isa(int* isa, int* aux,int* sa, int n) {

        int index = blockIdx.x * blockDim.x + threadIdx.x;
        int stride = blockDim.x * gridDim.x;

        for (int i = index; i < n; i+=stride) {
            isa[sa[i]]=aux[i];
        }
    }

    template <typename Comp>
    void initialize_isa(int* isa, int* sa, int* aux, int n, Comp comp) {

        fill_aux_for_isa<<<NUM_BLOCKS,NUM_THREADS_PER_BLOCK>>>(sa,aux,n, comp);

        hipDeviceSynchronize();

        Max_without_branching max;

        prefix_sum_cub_inclusive(aux,max, n);

        hipDeviceSynchronize();

        scatter_to_isa<<<NUM_BLOCKS,NUM_THREADS_PER_BLOCK>>>(isa,aux,sa,n);

        hipDeviceSynchronize();

    }


    static void prefix_doubling_gpu(const char* text, int* out_sa, int n) {
        
        char* gpu_text;
        int* sa;
        //Wofür??
        int* isa_container;
        int* aux_container;


        hipMallocManaged(&gpu_text, n*sizeof(char));

        hipMallocManaged(&sa, n*sizeof(int));
        hipMallocManaged(&isa_container, n*sizeof(int));
        hipMallocManaged(&aux_container, n*sizeof(int));

        //Copy text to GPU
        memset(gpu_text, 0, n*sizeof(char));
        hipMemcpy(gpu_text, text, n*sizeof(char), hipMemcpyHostToDevice);

        hipDeviceSynchronize();

        initialize_sa_gpu<<<NUM_BLOCKS,NUM_THREADS_PER_BLOCK>>>(n, sa);


        hipDeviceSynchronize();


        inital_sorting(gpu_text, sa, aux_container, n);

        hipDeviceSynchronize();

        copy_to_array<<<NUM_BLOCKS,NUM_THREADS_PER_BLOCK>>>(sa,aux_container,n);
        hipDeviceSynchronize();


        Compare_first_char comp(gpu_text);

        initialize_isa(out_sa,sa,aux_container,n, comp);

        hipDeviceSynchronize();

        std::cout<<std::endl;
        std::cout<<"ISA: ";
        for(int i = 0 ; i< n ; ++i) {
            std::cout<<out_sa[sa[i]]<<", ";
        }
        std::cout<<std::endl;

        /*
        std::cout<<"Hallo"<<std::endl;
        for(int index = 0; index < n; ++index) {
            std::cout<<sa[index]<<", ";
        }
        std::cout<<std::endl;
        */

        //int h = 4;
        // Sort by h characters
        //compare_first_four_chars cmp_init = compare_first_four_chars(text);

        
        //Initiale Sortierung
        //Möglichkeit 1: mit Thrust sortieren > direkt mit Key Funktion nutzbar aber langsamer als CUB -> Thrust Vectoren benötigt, meh
        //Möglichkeit 2: CUB nach nur einem Buchstaben -> Meh
        //Möglichkeit 3: CUB, aber vorher den Text mittels wordpacking transformieren
        //util::sort::ips4o_sort_parallel(sa, cmp_init);
/*      initialize_isa<sa_index, compare_first_four_chars>(sa, isa, aux,
                                                           cmp_init);
        phase.split("Mark singletons");
        mark_singletons(sa, isa);
        phase.split("Loop Initialization");

        // std::cout << "isa: " << isa << std::endl;
        size_t size = sa.size();
        size_t s = 0;

        auto tuple_container =
            util::make_container<std::tuple<sa_index, sa_index, sa_index>>(
                size);
        util::span<std::tuple<sa_index, sa_index, sa_index>> tuples;
        compare_tuples<sa_index> cmp;
        while (size > 0) {
            phase.split("Iteration");
            aux = util::span<sa_index>(aux_container).slice(0, size);
            tuples = tuple_container.slice(0, size);

            //s = create_tuples<sa_index>(tuples.slice(0, size), size, h, sa, isa);
            s = create_tuples_parallel<sa_index>(tuples.slice(0, size),
                    size, h, sa, isa, aux);
            //std::cout << "Elements left: " << size << std::endl;

            // std::cout << "Next size: " << s << std::endl;
            // Skip all operations till size gets its new size, if this
            // iteration contains no tuples
            if (s > 0) {
                tuples = tuples.slice(0, s);
                aux = util::span<sa_index>(aux).slice(0, s);
                // std::cout << "Sorting tuples." << std::endl;
                cmp = compare_tuples<sa_index>(tuples);
                util::sort::std_par_stable_sort(tuples, cmp);
                sa = sa.slice(0, s);
                update_ranks_prefixsum(tuples, aux);
                // std::cout << "Writing new order to sa." << std::endl;
                for (size_t i = 0; i < s; ++i) {
                    sa[i] = std::get<0>(tuples[i]);
                }

                for (size_t i = 0; i < s; ++i) {

                    isa[std::get<0>(tuples[i])] =
                        aux[i]; // std::get<1>(tuples[i]);
                }
                mark_singletons(sa, isa);
            }
            size = s;
            h = 2 * h;
        }
        phase.split("Write out SA");
        for (size_t i = 0; i < out_sa.size(); ++i) {
            out_sa[isa[i] ^ utils<sa_index>::NEGATIVE_MASK] = i;
        }
        */
    }
int main()
{
    std::string text_str = "caabaccaabacaa";
    const char* text = text_str.c_str();
    int n = text_str.size()+1;
    std::cout<<"n: "<<n<<std::endl;

    int* out_sa;
    hipMallocManaged(&out_sa, n*sizeof(int));

    prefix_doubling_gpu(text, out_sa, n);
    return 0;
}
