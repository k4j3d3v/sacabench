#include<iostream>
#include<cstdint>
#include<cstddef>

#include "cuda_util.cuh"

#define map_single_error_code(e) case e: out << #e; break;

static void map_error_code(hipError_t e, std::ostream& out) {
    switch (e) {
    map_single_error_code(hipErrorMissingConfiguration)
    map_single_error_code(hipErrorOutOfMemory)
    map_single_error_code(hipErrorNotInitialized)
    map_single_error_code(hipErrorLaunchFailure)
    map_single_error_code(hipErrorPriorLaunchFailure)
    map_single_error_code(hipErrorLaunchTimeOut)
    map_single_error_code(hipErrorLaunchOutOfResources)
    map_single_error_code(hipErrorInvalidDeviceFunction)
    map_single_error_code(hipErrorInvalidConfiguration)
    map_single_error_code(hipErrorInvalidDevice)
    map_single_error_code(hipErrorInvalidValue)
    map_single_error_code(hipErrorInvalidPitchValue)
    map_single_error_code(hipErrorInvalidSymbol)
    map_single_error_code(hipErrorMapFailed)
    map_single_error_code(hipErrorUnmapFailed)
    map_single_error_code(cudaErrorInvalidHostPointer)
    map_single_error_code(hipErrorInvalidDevicePointer)
    map_single_error_code(hipErrorInvalidTexture)
    map_single_error_code(cudaErrorInvalidTextureBinding)
    map_single_error_code(hipErrorInvalidChannelDescriptor)
    map_single_error_code(hipErrorInvalidMemcpyDirection)
    map_single_error_code(cudaErrorAddressOfConstant)
    map_single_error_code(cudaErrorTextureFetchFailed)
    map_single_error_code(cudaErrorTextureNotBound)
    map_single_error_code(cudaErrorSynchronizationError)
    map_single_error_code(cudaErrorInvalidFilterSetting)
    map_single_error_code(cudaErrorInvalidNormSetting)
    map_single_error_code(cudaErrorMixedDeviceExecution)
    map_single_error_code(hipErrorDeinitialized)
    map_single_error_code(hipErrorUnknown)
    map_single_error_code(cudaErrorNotYetImplemented)
    map_single_error_code(cudaErrorMemoryValueTooLarge)
    map_single_error_code(hipErrorInvalidHandle)
    map_single_error_code(hipErrorNotReady)
    map_single_error_code(hipErrorInsufficientDriver)
    map_single_error_code(hipErrorSetOnActiveProcess)
    map_single_error_code(cudaErrorInvalidSurface)
    map_single_error_code(hipErrorNoDevice)
    map_single_error_code(hipErrorECCNotCorrectable)
    map_single_error_code(hipErrorSharedObjectSymbolNotFound)
    map_single_error_code(hipErrorSharedObjectInitFailed)
    map_single_error_code(hipErrorUnsupportedLimit)
    map_single_error_code(cudaErrorDuplicateVariableName)
    map_single_error_code(cudaErrorDuplicateTextureName)
    map_single_error_code(cudaErrorDuplicateSurfaceName)
    map_single_error_code(cudaErrorDevicesUnavailable)
    map_single_error_code(hipErrorInvalidImage)
    map_single_error_code(hipErrorNoBinaryForGpu)
    map_single_error_code(cudaErrorIncompatibleDriverContext)
    map_single_error_code(hipErrorPeerAccessAlreadyEnabled)
    map_single_error_code(hipErrorPeerAccessNotEnabled)
    map_single_error_code(hipErrorContextAlreadyInUse)
    map_single_error_code(hipErrorProfilerDisabled)
    map_single_error_code(hipErrorProfilerNotInitialized)
    map_single_error_code(hipErrorProfilerAlreadyStarted)
    map_single_error_code(hipErrorProfilerAlreadyStopped)
    map_single_error_code(cudaErrorStartupFailure)
    map_single_error_code(cudaErrorApiFailureBase)
    default:
        out << "unknown code " << int(e);
    }
}

void cuda_check_internal(char const* file, int line, hipError_t v, char const* reason) {
    if (v != hipSuccess) {
        std::cerr << "CUDA ERROR at " << file << ":" << line << ": ";
        map_error_code(v, std::cerr);
        if (std::string(reason) != "") {
            std::cerr << " (" << reason << ")" << std::endl;
        }
        std::abort();
    }
}
