/*******************************************************************************
 * Copyright (C) 2019 Hermann Foot <hermann.foot@tu-dortmund.de>
 *
 * All rights reserved. Published under the BSD-3 license in the LICENSE file.
 ******************************************************************************/

#include<hip/hip_runtime.h>
#include "cuda_wrapper_interface.hpp"
#include "cuda_util.cuh"

#include "check_for_gpu_interface.hpp"
#include <iostream>

int cuda_GPU_available(){
    int deviceCount;
    int deviceNumber;
    size_t free_bytes;
    size_t total_bytes;
    hipError_t e1 = hipGetDeviceCount(&deviceCount);
    hipError_t e2 = hipGetDevice(&deviceNumber);   
    hipError_t e3=hipMemGetInfo(&free_bytes, &total_bytes);
    bool available = (e1 == hipSuccess) && (e2 == hipSuccess) && (e3 == hipSuccess);
    return available;
}