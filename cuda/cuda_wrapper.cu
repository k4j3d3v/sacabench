#include<iostream>
#include<cstdint>
#include<cstddef>

#include <hip/hip_runtime.h>
#include "../external/cub/hipcub/hipcub.hpp"

#include "cuda_wrapper_interface.hpp"

void cuda_check(bool v, char const* reason) {
    if (!v) {
        std::cerr << "cuda error: " << reason << std::endl;
    }
    std::abort();
}

void* allocate_cuda_buffer(size_t size) {
    void* ret = nullptr;
    cuda_check(hipMalloc(&ret, size), "hipMalloc");
    return ret;
}

void* allocate_managed_cuda_buffer(size_t size) {
    void* ret = nullptr;
    cuda_check(hipMallocManaged(&ret, size), "hipMallocManaged");
    return ret;
}

void free_cuda_buffer(void* ptr) {
    cuda_check(hipFree(ptr), "hipFree");
}

void exclusive_sum_64(uint64_t* d_in,
                      uint64_t* d_out,
                      size_t num_items)
{
    //using hipcub::DeviceScan::ExclusiveSum;

    // Determine temporary device storage requirements
    size_t temp_storage_bytes = 0;
    cuda_check(
        hipcub::DeviceScan::ExclusiveSum(nullptr,
                                      temp_storage_bytes,
                                      d_in,
                                      d_out,
                                      num_items), "ExclusiveSum");

    // Allocate temporary storage
    void* d_temp_storage = allocate_managed_cuda_buffer(temp_storage_bytes);

    // Run prefix sum
    cuda_check(
        hipcub::DeviceScan::ExclusiveSum(d_temp_storage,
                                      temp_storage_bytes,
                                      d_in,
                                      d_out,
                                      num_items), "ExclusiveSum");

    hipDeviceSynchronize();

    free_cuda_buffer(d_temp_storage);
}
