#include "hip/hip_runtime.h"
#include <string>
#include <iostream>
#include "cub-1.8.0/hipcub/hipcub.hpp"

template <uint32_t BLOCK_THREADS, uint32_t ITEMS_PER_THREAD>
__global__ static void mark_heads(uint32_t* d_in, bool* d_flags) {

    // Specialize BlockDiscontinuity for a 1D block of 128 threads on type int
    typedef hipcub::BlockDiscontinuity<uint32_t, ITEMS_PER_THREAD> BlockDiscontinuity;
    // Allocate shared memory for BlockDiscontinuity
    __shared__ typename BlockDiscontinuity::TempStorage temp_storage;

    // Specialize BlockLoad type for our thread block
    typedef hipcub::BlockLoad<uint32_t, BLOCK_THREADS, ITEMS_PER_THREAD, hipcub::BLOCK_LOAD_WARP_TRANSPOSE> BlockLoadT;
    // Specialize BlockStore type for our thread block
    typedef hipcub::BlockStore<bool, BLOCK_THREADS, ITEMS_PER_THREAD, hipcub::BLOCK_STORE_WARP_TRANSPOSE> BlockStoreT;

    // Shared memory
    __shared__ typename BlockLoadT::TempStorage    load;
    __shared__ typename BlockStoreT::TempStorage   store;

    // Obtain a segment of consecutive items that are blocked across threads
    uint32_t thread_data[ITEMS_PER_THREAD];
    BlockLoadT(load).Load(d_in, thread_data);

    // block of flags per thread
    bool flags[4];
    // Collectively compute head flags for discontinuities in the segment
    BlockDiscontinuity(temp_storage).FlagHeads(flags, thread_data, hipcub::Inequality());

    // Store flags from a blocked arrangement
    BlockStoreT(store).Store(d_flags, flags);
}

int main()
{
    const uint32_t g_grid_size = 1; // WTF ist this???
    const uint32_t BLOCK_THREADS = 1024;
    const uint32_t ITEMS_PER_THREAD = 4;
    const int TILE_SIZE = BLOCK_THREADS * ITEMS_PER_THREAD;

    uint32_t* h_sa = new uint32_t[TILE_SIZE];
    bool* h_head_flags = new bool[TILE_SIZE];

    for (uint32_t i = 0; i < TILE_SIZE; ++i) {
        h_sa[i] = (i/2) % 10;
    }

    uint32_t* d_sa = NULL;
    bool* d_head_flags = NULL;

    hipMalloc((void**)&d_sa, sizeof(uint32_t) * TILE_SIZE);
    hipMalloc((void**)&d_head_flags, sizeof(bool) * TILE_SIZE);

    for (uint32_t i = 0; i < TILE_SIZE; ++i) {
        std::cout << h_sa[i] << " ";
        h_head_flags[i] = false;
    }
    std::cout << std::endl;

    // copy problem to device
    hipMemcpy(d_sa, h_sa, sizeof(uint32_t) * TILE_SIZE, hipMemcpyHostToDevice);

    hipDeviceSynchronize();

    mark_heads<BLOCK_THREADS, ITEMS_PER_THREAD><<<g_grid_size, BLOCK_THREADS>>>(d_sa, d_head_flags);

    hipDeviceSynchronize();

    // copy solution to host
    hipMemcpy(h_head_flags, d_head_flags, sizeof(bool) * TILE_SIZE, hipMemcpyDeviceToHost);

    for (uint32_t i = 0; i < TILE_SIZE; ++i) {
        std::cout << h_head_flags[i] << " ";
    }
    std::cout << std::endl;

    if(h_sa) delete[] h_sa;
    if(h_head_flags) delete[] h_head_flags;
    if(d_sa) hipFree(d_sa);
    if(d_head_flags) hipFree(d_head_flags);

    return 0;
}
